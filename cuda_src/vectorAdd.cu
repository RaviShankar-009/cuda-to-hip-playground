#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <string>

__global__ void vadd(const float* __restrict__ a,
                     const float* __restrict__ b,
                     float* __restrict__ c,
                     size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) c[i] = a[i] + b[i];
}

int main(int argc, char** argv) {
    size_t n = (argc > 1) ? std::stoull(std::string(argv[1])) : (1ull << 24); // ~16M
    size_t bytes = n * sizeof(float);

    float *ha = (float*)std::malloc(bytes);
    float *hb = (float*)std::malloc(bytes);
    float *hc = (float*)std::malloc(bytes);
    if (!ha || !hb || !hc) { std::fprintf(stderr, "Host alloc failed\n"); return 1; }
    for (size_t i = 0; i < n; ++i) { ha[i] = float(i) * 0.001f; hb[i] = 1.0f; }

    float *da = nullptr, *db = nullptr, *dc = nullptr;
    hipMalloc(&da, bytes);
    hipMalloc(&db, bytes);
    hipMalloc(&dc, bytes);

    hipMemcpy(da, ha, bytes, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, bytes, hipMemcpyHostToDevice);

    int block = 256;
    int grid  = int((n + block - 1) / block);

    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    vadd<<<grid, block>>>(da, db, dc, n);
    hipEventRecord(stop); hipEventSynchronize(stop);

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);

    hipMemcpy(hc, dc, bytes, hipMemcpyDeviceToHost);

    // Verify a few elements
    for (size_t i = 0; i < 10; ++i) {
        float ref = ha[i] + hb[i];
        if (std::fabs(hc[i] - ref) > 1e-4f) {
            std::fprintf(stderr, "Mismatch at %zu: got %f, want %f\n", i, hc[i], ref);
            break;
        }
    }

    double gb   = 3.0 * double(bytes) / 1e9; // read A,B + write C
    double gbps = gb / (ms / 1e3);
    std::printf("CUDA vadd: N=%zu time=%.3f ms, BW=%.2f GB/s\n", n, ms, gbps);

    hipFree(da); hipFree(db); hipFree(dc);
    std::free(ha); std::free(hb); std::free(hc);
    return 0;
}
